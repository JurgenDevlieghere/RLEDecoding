#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdlib.h>

#include <sys/time.h>


#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#define N 1000
#define NBLOCKS 1
#define BLOCK_SIZE 1024

// a function that calculates the time difference in msec
unsigned int TimeDiff(struct timespec start, struct timespec end)
{
    return (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
}


unsigned int FillWithRandomNumbers(int* values, unsigned char* lengths, int n)
{
    unsigned int totalLength=0;

    srand((unsigned)time(0));
    for (int i=0;i<N;i++)
    {
        *values=rand() % 0xffffffff;
        unsigned char length=rand() & 0xff;
        *lengths = length;
        totalLength+=(unsigned int)length;
        values++;
        lengths++;
    }

    return totalLength; // the decoded mesage will need this much of space.
}

__global__ void decodeRLE(int* values, unsigned char * lengths, unsigned int* np, unsigned int * output)
{
    unsigned int n=*np;
    extern __shared__ unsigned int sumOfLengths[];

    cg::grid_group grid=cg::this_grid();
    int threadId=blockIdx.x * blockDim.x + threadIdx.x;

    /// we have only half of the threads. each take care of the odd and even.
    int evenIndex=threadId*2;

    // step 0
    sumOfLengths[threadId]=(threadId==0 ? 0 : lengths[evenIndex-1])+lengths[evenIndex]; 
    grid.sync();

    // steps 1 .. n-1
    for (unsigned int groupSize=1, twiceGroupSize=2;twiceGroupSize<n;groupSize=twiceGroupSize, twiceGroupSize*=2)
    {
        unsigned int groupNr=threadId/twiceGroupSize;
        unsigned int minThreadId=groupSize*(1+groupNr*2);
        unsigned int maxThreadId=minThreadId+groupSize-1;
        if (threadId>=minThreadId && threadId<=maxThreadId)
        {
            unsigned int toAdd=sumOfLengths[minThreadId-1]; 
            sumOfLengths[threadId]+=toAdd;
        }
        grid.sync();
    }

    // last step. Fill even.
    int bothOffsets[2];
    bothOffsets[0]=threadId==0 ? 0 : sumOfLengths[threadId-1]+lengths[evenIndex-1];
    bothOffsets[1]=sumOfLengths[threadId];

    grid.sync();

    // now do real decompression and place everything in output buffer. Do for both indexes
    for (int index=0;index<=1;index++) // just 2 passes
    {
        for (int i=bothOffsets[index];i<bothOffsets[index]+(unsigned int)lengths[evenIndex+index];i++)
        {
            output[i]=values[evenIndex+index];
        }    
    }
}

#define NLOOPS 20


int main()
{
    int *values;
    unsigned int *n;
    unsigned char *lengths;

    hipMallocManaged(&n,sizeof(unsigned int)); // n contains the size of the RLE sequence
    hipMallocManaged(&values,N*sizeof(int));
    hipMallocManaged(&lengths,N*sizeof(unsigned char));

    *n=N;
    // create a long rle sequence
    unsigned int totalLength=FillWithRandomNumbers(values, lengths, N);

    printf("total length of exploded RLE: %ld\n", totalLength);

    // the place where the decompressed message should end up
    unsigned int* output;
    hipMallocManaged(&output,totalLength*sizeof(unsigned int));

    hipDeviceSynchronize();

    void *kernelArgs[] = 
    {
        (void*) &values, (void *)&lengths, (void*)&n, (void *) &output
    };

    // start of the real work
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    
    for (int count=0;count<NLOOPS;count++)
    {

        hipLaunchCooperativeKernel((void*)decodeRLE, // function
                                        1, // grid dimensions
                                        N/2, // block dimensions
                                        kernelArgs, 
                                        N/2*sizeof(unsigned int) // shared memory. Unsigned ints because summing multiple unsigned chars > 255
                                    );
        hipDeviceSynchronize();   
    }

    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    unsigned int timeElapsed = TimeDiff(start, end); // in microseconds
    unsigned int timePerRLE = timeElapsed / NLOOPS;
    printf("time per RLE %ld microsecs\n", timePerRLE);

}

